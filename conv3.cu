#include "hip/hip_runtime.h"
/*
 * conv3.cu
 *
 *  Created on: Nov 15, 2009
 *      Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 */
#include <math.h>
#include <nvmatrix.cuh>
#include "conv3.cuh"

void _convolve3_bw(float* images, float* filters, float* targets, int numCases,
                  int numFilters, int imgSize, int filterSize, int stride, bool useDynamics = false) {
    assert(stride == 1 || stride == 3);
    int numOutputsX = imgSize - filterSize + 1;
//    int numOutputs = numOutputsX*numOutputsX;
    bool checkOutputBounds = numOutputsX % 16 != 0;
    if(filterSize > 37) {
        int numPartsX = DIVUP(numOutputsX, 16);
        int numParts = numPartsX*numPartsX;
        int blocksY = numParts, blocksX = numCases;
        dim3 grid(blocksX, blocksY);
        dim3 threads(16, 16);
        bool checkFilterBounds = filterSize % 16 != 0;
//        printf("check filter bounds: %d, check output bounds: %d, stride: %d\n", checkFilterBounds, checkOutputBounds, stride);
        if(checkFilterBounds) {
            if (checkOutputBounds) {
                if (stride == 1) {
                    conv3_bw_nofit_16x16<true, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters, filterSize);
                } else {
                    conv3_bw_nofit_16x16<true, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters, filterSize);
                }
            } else {
                if (stride == 1) {
                    conv3_bw_nofit_16x16<false, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters, filterSize);
                } else {
                    conv3_bw_nofit_16x16<false, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters, filterSize);
                }
            }
        } else {
            if (checkOutputBounds) {
                if (stride == 1) {
                    conv3_bw_nofit_16x16<true, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters, filterSize);
                } else {
                    conv3_bw_nofit_16x16<true, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters, filterSize);
                }
            } else {
                if (stride == 1) {
                    conv3_bw_nofit_16x16<false, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters, filterSize);
                } else {
                    conv3_bw_nofit_16x16<false, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters, filterSize);
                }
            }
        }
    } else {
        if (useDynamics) {
            // later
        } else {
            int numPartsX = DIVUP(numOutputsX, 16);
            int numParts = numPartsX*numPartsX;
            int blocksY = numParts, blocksX = numCases;
            dim3 grid(blocksX, blocksY);
            dim3 threads(16, 16);

            /*
             * This code was auto-generated...
             */
            if (filterSize == 1) {
                throw "try multByScalar";
            } else if (filterSize == 2) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<2, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<2, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<2, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<2, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 3) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<3, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<3, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<3, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<3, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 4) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<4, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<4, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<4, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<4, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 5) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<5, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<5, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<5, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<5, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 6) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<6, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<6, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<6, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<6, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 7) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<7, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<7, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<7, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<7, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 8) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<8, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<8, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<8, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<8, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 9) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<9, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<9, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<9, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<9, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 10) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<10, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<10, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<10, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<10, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 11) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<11, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<11, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<11, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<11, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 12) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<12, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<12, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<12, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<12, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 13) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<13, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<13, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<13, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<13, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 14) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<14, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<14, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<14, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<14, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 15) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<15, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<15, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<15, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<15, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 16) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<16, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<16, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<16, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<16, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 17) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<17, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<17, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<17, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<17, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 18) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<18, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<18, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<18, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<18, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 19) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<19, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<19, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<19, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<19, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 20) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<20, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<20, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<20, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<20, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 21) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<21, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<21, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<21, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<21, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 22) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<22, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<22, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<22, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<22, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 23) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<23, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<23, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<23, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<23, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 24) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<24, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<24, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<24, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<24, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 25) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<25, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<25, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<25, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<25, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 26) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<26, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<26, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<26, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<26, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 27) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<27, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<27, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<27, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<27, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 28) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<28, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<28, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<28, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<28, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 29) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<29, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<29, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<29, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<29, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 30) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<30, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<30, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<30, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<30, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 31) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<31, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<31, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<31, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<31, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 32) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<32, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<32, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<32, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<32, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 33) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<33, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<33, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<33, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<33, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 34) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<34, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<34, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<34, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<34, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 35) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<35, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<35, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<35, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<35, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 36) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<36, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<36, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<36, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<36, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }  else if (filterSize == 37) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<37, true, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<37, true, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<37, false, 1><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    } else {
                        conv3_bw_fit_16x16<37, false, 3><<<grid, threads>>>(images, filters, targets, imgSize, numFilters);
                    }
                }
            }
        }
    }
    cutilCheckMsg("kernel execution failed");
}

void convolve3_bw(NVMatrix* images, NVMatrix* filters, NVMatrix* targets) {
    int numFilters = filters->getNumRows();
    int numCases = images->getNumRows();
    assert(images->getNumCols() % numFilters == 0);
//    assert(filters->getNumCols() % numFilters == 0);
    int imgPixels = images->getNumCols() / numFilters;
    int filterPixels = filters->getNumCols();
    assert(sqrt(double(imgPixels)) == floor(sqrt(double(imgPixels))));
    assert(sqrt(double(filterPixels)) == floor(sqrt(double(filterPixels))));
    int imgSize = int(sqrt(double(imgPixels)));
    int filterSize = int(sqrt(double(filterPixels)));

    int numOutputsX = imgSize - filterSize + 1;
    int numOutputs = numOutputsX * numOutputsX;

    assert(targets->getNumElements() == numOutputs * numCases);
    assert(!images->isTrans());
    assert(!filters->isTrans());
    assert(!targets->isTrans());
    assert(imgSize > filterSize);

//    printf("computed numcases: %d, numfilters: %d, imgsize: %d, filtersize: %d\n", numCases, numFilters, imgSize, filterSize);
    _convolve3_bw(images->getDevData(), filters->getDevData(), targets->getDevData(),
                 numCases, numFilters, imgSize, filterSize, 1);
}

void convolve3_color(NVMatrix* images, NVMatrix* filters, NVMatrix* targets) {
    int numFilters = filters->getNumRows();
    int numCases = images->getNumRows();
    assert(images->getNumCols() % numFilters == 0);
//    assert(filters->getNumCols() % numFilters == 0);
    int imgPixels = images->getNumCols() / numFilters;
    assert(filters->getNumCols() % 3 == 0);
    int filterPixels = filters->getNumCols() / 3;
    assert(sqrt(double(imgPixels)) == floor(sqrt(double(imgPixels))));
    assert(sqrt(double(filterPixels)) == floor(sqrt(double(filterPixels))));
    int imgSize = int(sqrt(double(imgPixels)));
    int filterSize = int(sqrt(double(filterPixels)));

    int numOutputsX = imgSize - filterSize + 1;
    int numOutputs = numOutputsX * numOutputsX;

    assert(targets->getNumElements() == 3*numOutputs * numCases);
    assert(!images->isTrans());
    assert(!filters->isTrans());
    assert(!targets->isTrans());
    assert(imgSize > filterSize);

//    printf("computed numcases: %d, numfilters: %d, imgsize: %d, filtersize: %d\n", numCases, numFilters, imgSize, filterSize);
    _convolve3_bw(images->getDevData(), filters->getDevData(), targets->getDevData(),
                 numCases*3, numFilters, imgSize, filterSize, 3);
}

