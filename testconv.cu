#include "hip/hip_runtime.h"
/*
 * testconv.cu
 *
 *  Created on: Oct 31, 2009
 *      Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 */

#include <cutil_inline.h>
#include <assert.h>
#include <nvmatrix.cuh>
#include <matrix.h>

#include "testconv_extras.cuh"
#include "conv.cuh"
#include "conv2.cuh"
#include "conv_util.cuh"
#include "conv3.cuh"
#include "convCPU.h"
#include "gpu_locking.h"

static uint timer;

void init_tests(int boardNum) {
    hipSetDevice(boardNum > -1 ? boardNum : cutGetMaxGflopsDeviceId());
    cublasInit();
    NVMatrix::initDeviceProps();
    NVMatrix::initRandom(7);
    cutilCheckError( cutCreateTimer( &timer));
}

void test_convolve(int imgSize, int filterSize, bool color) {
    printf("===============================\n");
    printf("test_convolve\n");
    printf("===============================\n");

    int numFilters = 64, numCases = 128;
    int filterPixels = filterSize * filterSize;
    int imgPixels = imgSize * imgSize;
    int numOutputsX = imgSize - filterSize + 1;
    int numOutputs = numOutputsX * numOutputsX;
    assert(numFilters % 8 == 0);
    printf("Images: %d, filters: %d\n", numCases, numFilters);
    printf("Image size: %dx%d, filter size: %dx%d\n", imgSize, imgSize, filterSize, filterSize);
    printf("Output grid: %dx%d\n", numOutputsX, numOutputsX);
    printf("Color: %s\n", color ? "yes" : "no");

    int colorMult = color ? 3 : 1;
    Matrix filters(numFilters, filterPixels * colorMult);
    Matrix images(numCases, imgPixels * colorMult);
    Matrix targets(numCases, numFilters * numOutputs);
    filters.randomizeUniform();
    images.randomizeUniform();
    targets.apply(Matrix::ZERO);

    NVMatrix nvFilters(filters, true);
    NVMatrix nvImages(images, true);
    NVMatrix nvTargets(targets, true); // eh why not

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));
    if(color) {
        convColorCPU(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numCases, numFilters);
    } else {
        convCPU(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numCases, numFilters);
    }
    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 6);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));
    if(color) {
        convolve_color(&nvImages, &nvFilters, &nvTargets);
    } else {
        convolve_bw(&nvImages, &nvFilters, &nvTargets);
    }
    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 6);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}

/*
 * This tests the routines in conv2.cuh. See the documentation there for an explanation.
 */
void test_convolve2(int imgSize, int filterSize, bool color) {
    printf("===============================\n");
    printf("test_convolve2\n");
    printf("===============================\n");

    int numFilters = 64, numCases = 128;
    int filterPixels = filterSize * filterSize;
    int imgPixels = imgSize * imgSize;
    int numOutputsX = imgSize - filterSize + 1;
    int numOutputs = numOutputsX * numOutputsX;
    assert(numFilters % 8 == 0);
    printf("Images: %d, filters: %d\n", numCases, numFilters);
    printf("Image size: %dx%d, filter size: %dx%d\n", imgSize, imgSize, filterSize, filterSize);
    printf("Output grid: %dx%d\n", numOutputsX, numOutputsX);
    printf("Color: %s\n", color ? "yes" : "no");

    int colorMult = color ? 3 : 1;
    Matrix filters(numCases, numFilters * filterPixels);
    Matrix images(numCases, imgPixels * colorMult);
    Matrix targets(numCases, numFilters * numOutputs * colorMult);
    filters.randomizeUniform();
    images.randomizeUniform();
    targets.apply(Matrix::ZERO);

    NVMatrix nvFilters(filters, true);
    NVMatrix nvImages(images, true);
    NVMatrix nvTargets(targets, true); // eh why not

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));
    if(color) {
        conv2ColorCPU(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numCases * 3, numFilters);
    } else {
        conv2CPU(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numCases, numFilters);
    }
    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 6);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));
    if(color) {
        convolve2_color(&nvImages, &nvFilters, &nvTargets,filterSize);
    } else {
        convolve2_bw(&nvImages, &nvFilters, &nvTargets, filterSize);
    }
    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 6);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}


void test_rot180(int filterSize, bool color) {
    printf("===============================\n");
    printf("test_rot180\n");
    printf("===============================\n");

    int numFilters = 64;
    int filterPixels = filterSize * filterSize;
    int colorMult = color ? 3 : 1;
    printf("Filters: %d\n",  numFilters);
    printf("Color: yes\n");

    Matrix filters(numFilters, colorMult * filterPixels);
    Matrix targets(filters);

    filters.randomizeUniform();

    targets.apply(Matrix::ZERO);

    NVMatrix nvFilters(filters, true);
    NVMatrix nvTargets(targets, true); // eh why not

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    rotate180CPU(filters.getData(), targets.getData(), filterSize, colorMult * numFilters);

    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 6);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    rotate180(&nvFilters, &nvTargets, color);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 6);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}

void test_padZeros(int imgSize, int paddingSize, bool color) {
    printf("===============================\n");
    printf("test_padZeros\n");
    printf("===============================\n");

    int numImages = 128;
    int imgPixels = imgSize * imgSize;
    int targetSize = imgSize + 2*paddingSize;
    int targetPixels = targetSize * targetSize;

    printf("Filters: %d\n",  numImages);
    printf("Color: yes\n");

    int colorMult = color ? 3 : 1;
    Matrix images(numImages, colorMult * imgPixels);
    Matrix targets(numImages, colorMult * targetPixels);

    images.randomizeUniform();

    targets.apply(Matrix::ZERO);

    NVMatrix nvImages(images, true);
    NVMatrix nvTargets(targets, true); // eh why not

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    padZerosCPU(images.getData(), targets.getData(), imgSize, colorMult * numImages, paddingSize);

    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 10);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    copyInto(&nvImages, &nvTargets, paddingSize, color);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 10);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}


/*
 * This tests the routines in conv3.cuh. See the documentation there for an explanation.
 */
void test_convolve3(int imgSize, int filterSize, bool color) {
    printf("===============================\n");
    printf("test_convolve3\n");
    printf("===============================\n");

    int numFilters = 64, numCases = 128;
    int filterPixels = filterSize * filterSize;
    int imgPixels = imgSize * imgSize;
    int numOutputsX = imgSize - filterSize + 1;
    int numOutputs = numOutputsX * numOutputsX;
    assert(numFilters % 8 == 0);
    printf("Images: %d, filters: %d\n", numCases, numFilters);
    printf("Image size: %dx%d, filter size: %dx%d\n", imgSize, imgSize, filterSize, filterSize);
    printf("Output grid: %dx%d\n", numOutputsX, numOutputsX);
    printf("Color: %s\n", color ? "yes" : "no");

    int colorMult = color ? 3 : 1;
    Matrix filters(numFilters, colorMult*filterPixels);
    Matrix images(numCases, numFilters * imgPixels);
    Matrix targets(numCases, numOutputs*colorMult);
    filters.randomizeUniform();
    images.randomizeUniform();
    targets.apply(Matrix::ZERO);

    NVMatrix nvFilters(filters, true);
    NVMatrix nvImages(images, true);
    NVMatrix nvTargets(targets, true); // eh why not

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));
    if(color) {
        conv3ColorCPU(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numCases, numFilters*3);
    } else {
        conv3CPU(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numCases, numFilters);
    }
    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 14, 6);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));
//    images.print(0,6,63*16,6);
    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));
    if(color) {
        convolve3_color(&nvImages, &nvFilters, &nvTargets);
    } else {
        convolve3_bw(&nvImages, &nvFilters, &nvTargets);
    }
    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 14, 6);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}

void test_subsample(int imgSize, int factor) {
    printf("===============================\n");
    printf("test_subsample\n");
    printf("===============================\n");

    assert(imgSize > factor && imgSize % factor == 0);
    int numImages = 128 * 64;
    int imgPixels = imgSize * imgSize;
    int numRegionsPerImage = (imgSize / factor)*(imgSize / factor);

    printf("Images: %d\n",  numImages);

    Matrix images(numImages, imgPixels);
    Matrix targets(numImages, numRegionsPerImage);

    images.randomizeUniform();

    targets.apply(Matrix::ZERO);

    NVMatrix nvImages(images, true);
    NVMatrix nvTargets(targets, true); // eh why not

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    subsampleCPU(images.getData(), targets.getData(), imgSize, factor, numImages);

    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 10);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    subsample(&nvImages, &nvTargets, factor);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 10);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}


void test_supersample(int imgSize, int factor, bool trans) {
    printf("===============================\n");
    printf("test_supersample\n");
    printf("===============================\n");

    int numImages = 128;
    int imgPixels = imgSize * imgSize;
    int targetPixels = imgPixels*factor*factor;

    printf("Images: %d\n",  numImages);
    printf("Image size: %dx%d\n", imgSize, imgSize);
    printf("Output size: %dx%d\n", imgSize*factor, imgSize*factor);

    NVMatrix nvImages(numImages, imgPixels, trans);
    NVMatrix nvTargets(numImages, targetPixels, false);

    nvImages.randomizeUniform();
    nvTargets.apply(NVMatrix::ZERO);

    Matrix images(numImages, imgPixels);
    Matrix targets(numImages, targetPixels);

    nvImages.copyToHost(images);
    targets.apply(Matrix::ZERO);

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    supersampleCPU(images.getData(), targets.getData(), imgSize, factor, numImages, trans);

    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 10);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    supersample(&nvImages, &nvTargets, factor);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 10);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}

void test_gridToMatrix(int imgSize, int squareSize) {
    printf("===============================\n");
    printf("test_gridToMatrix\n");
    printf("===============================\n");
    assert(imgSize % squareSize == 0);
    int numImages = 128*96;
    int imgPixels = imgSize * imgSize;
    int regionsPerImage = (imgSize / squareSize) * (imgSize / squareSize);

    printf("Images: %d\n",  numImages);
    printf("Image size: %dx%d\n", imgSize, imgSize);
    printf("Square size: %dx%d\n", squareSize, squareSize);
    printf("Output matrix: %dx%d\n", numImages * squareSize * regionsPerImage, squareSize);

    NVMatrix nvImages(numImages, imgPixels, false);
    NVMatrix nvTargets(numImages * regionsPerImage, squareSize * squareSize, false);

    nvImages.randomizeUniform();
    nvTargets.apply(NVMatrix::ZERO);

    Matrix images(numImages, imgPixels);
    Matrix targets(numImages * regionsPerImage, squareSize * squareSize);

    nvImages.copyToHost(images);
    targets.apply(Matrix::ZERO);

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    gridToMatrixCPU(images.getData(), targets.getData(), imgSize, squareSize, numImages);

    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 10);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    gridToMatrix(&nvImages, &nvTargets, squareSize, true);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 10);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}


void test_matrixToGrid(int imgSize, int squareSize) {
    printf("===============================\n");
    printf("test_matrixToGrid\n");
    printf("===============================\n");
    assert(imgSize % squareSize == 0);
    int numImages = 128*96;
    int imgPixels = imgSize * imgSize;
    int regionsPerImage = (imgSize / squareSize) * (imgSize / squareSize);

    printf("Images: %d\n",  numImages);
    printf("Image size: %dx%d\n", imgSize, imgSize);
    printf("Square size: %dx%d\n", squareSize, squareSize);
    printf("Output matrix: %dx%d\n", numImages, imgPixels);

    NVMatrix nvImages(numImages * regionsPerImage, squareSize * squareSize, false);
    NVMatrix nvTargets(numImages, imgPixels, false);

    nvImages.randomizeUniform();
    nvTargets.apply(NVMatrix::ZERO);

    Matrix images(numImages * regionsPerImage, squareSize * squareSize);
    Matrix targets(numImages, imgPixels);

    nvImages.copyToHost(images);
    targets.apply(Matrix::ZERO);

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    matrixToGridCPU(images.getData(), targets.getData(), imgSize, squareSize, numImages);

    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 10);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    matrixToGrid(&nvImages, &nvTargets, squareSize, true);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 10);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}

void test_localMax(int imgSize, int squareSize) {
    printf("===============================\n");
    printf("test_localMax\n");
    printf("===============================\n");
    assert(imgSize % squareSize == 0);
    int numImages = 128*64;
    int imgPixels = imgSize * imgSize;
    int regionsPerImage = (imgSize / squareSize) * (imgSize / squareSize);

    NVMatrix nvImages(numImages, imgPixels, false);
    NVMatrix nvTargets(numImages * regionsPerImage, squareSize * squareSize, false);
    NVMatrix nvTargetsMax(numImages * regionsPerImage, 1, false);

    nvImages.randomizeUniform();
    nvTargets.apply(NVMatrix::ZERO);
    nvTargetsMax.apply(NVMatrix::ZERO);

    Matrix images(numImages, imgPixels);
    Matrix targets(numImages * regionsPerImage, squareSize * squareSize);
    Matrix targetsSum(numImages * regionsPerImage, 1);

    printf("Images: %d\n",  numImages);
    printf("Image size: %dx%d\n", imgSize, imgSize);
    printf("Square size: %dx%d\n", squareSize, squareSize);
    printf("Output matrix: %dx%d\n", nvTargets.getNumRows(), nvTargets.getNumCols());

    nvImages.copyToHost(images);
    targets.apply(Matrix::ZERO);

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    gridToMatrixCPU(images.getData(), targets.getData(), imgSize, squareSize, numImages);
    targets.sum(1, targetsSum);
    targets.eltWiseDivideByVector(targetsSum);
    matrixToGridCPU(targets.getData(), images.getData(), imgSize, squareSize, numImages);
    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 10);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    gridToMatrix(&nvImages, &nvTargets, squareSize, true);
    nvTargets.max(1, nvTargetsMax);
    nvTargets.eltWiseDivideByVector2(nvTargetsMax);
    matrixToGrid(&nvTargets, &nvImages, squareSize, true);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 10);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}

void test_localSum(int imgSize, int squareSize) {
    printf("===============================\n");
    printf("test_localSum\n");
    printf("===============================\n");
    assert(imgSize % squareSize == 0);
    int numImages = 128*64;
    int imgPixels = imgSize * imgSize;
    int regionsPerImage = (imgSize / squareSize) * (imgSize / squareSize);

    NVMatrix nvImages(numImages, imgPixels, false);
    NVMatrix nvTargets(numImages * regionsPerImage, squareSize * squareSize, false);
    NVMatrix nvTargetsSum(numImages * regionsPerImage, 1, false);

    nvImages.randomizeUniform();
    nvTargets.apply(NVMatrix::ZERO);
    nvTargetsSum.apply(NVMatrix::ZERO);

    Matrix images(numImages, imgPixels);
    Matrix targets(numImages * regionsPerImage, squareSize * squareSize);
    Matrix targetsSum(numImages * regionsPerImage, 1);

    printf("Images: %d\n",  numImages);
    printf("Image size: %dx%d\n", imgSize, imgSize);
    printf("Square size: %dx%d\n", squareSize, squareSize);
    printf("Output matrix: %dx%d\n", nvTargets.getNumRows(), nvTargets.getNumCols());

    nvImages.copyToHost(images);
    targets.apply(Matrix::ZERO);

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    gridToMatrixCPU(images.getData(), targets.getData(), imgSize, squareSize, numImages);
    targets.sum(1, targetsSum);
    targets.eltWiseDivideByVector(targetsSum);
    matrixToGridCPU(targets.getData(), images.getData(), imgSize, squareSize, numImages);
    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 10);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    gridToMatrix(&nvImages, &nvTargets, squareSize, true);
    nvTargets.sum(1, nvTargetsSum);
    nvTargets.eltWiseDivideByVector2(nvTargetsSum);
    matrixToGrid(&nvTargets, &nvImages, squareSize, true);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 10);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}

void test_sampleMultinomial(int nomials) {
    printf("===============================\n");
    printf("test_sampleMultinomial\n");
    printf("===============================\n");
    int multinomials = 128*25*96;

    Matrix multi(multinomials, nomials);
    Matrix randoms(multinomials,1);
    Matrix targets(multi);

    multi.randomizeUniform();
    Matrix& multiSum = multi.sum(1);
    multiSum.addScalar(1); // this will make "none of the above" an option
    multi.eltWiseDivideByVector(multiSum);
//    multi.print(3,16);
    randoms.randomizeUniform();
    targets.apply(Matrix::ZERO);

    NVMatrix nvMulti(multi, true);
    NVMatrix nvRandoms(randoms, true);
    NVMatrix nvTargets(targets, true);

    printf("Multinomial distributions: %d\n",  multinomials);
    printf("Multinomial distribution size: %d\n", nomials);

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));

    sampleMultinomialCPU(multi.getData(), randoms.getData(), targets.getData(),multinomials, nomials);

    cutilCheckError( cutStopTimer( timer));
    printf("CPU (partial) result:\n");
    targets.print(32*16-3, 6, 0, 8);
    printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    cutilCheckError( cutResetTimer( timer));
    cutilCheckError( cutStartTimer( timer));
//    nvMulti.print(3,16);
    sampleMultinomial(&nvMulti, &nvRandoms, &nvTargets);

    hipDeviceSynchronize();
    cutilCheckError( cutStopTimer( timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(32*16-3, 6, 0, 8);
    printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);

    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    Matrix &s = cpuNVTargets.sum(0);
    printf("Number of distributions sampled differently: %d (this may be non-zero, but only slightly)\n", int(s.sum())/2);
}

int main(int argc, char** argv) {
    // This line just for compiling and examining profiler output.
//    exit(0); conv2_bw_nofit_dynXYZ_2per<true, false,3,8,8><<<1,1>>>(NULL, NULL, NULL, 0,0, 0);
//    exit(0); conv_bw_fit_4x16_2per<8,true,3><<<1,1>>>(NULL, NULL, NULL, 0);
//    exit(0); kSampleSmallMultinomial<15,16><<<1,1>>>(NULL, NULL, NULL, 0, 0);
    int boardNum = get_board_lock();
    if (boardNum == GPU_LOCK_NO_BOARD) {
        printf("No free GPU boards!\n");
        exit(EXIT_FAILURE);
    } else if(boardNum == GPU_LOCK_NO_SCRIPT) {
        printf("Running on default board.\n");
    } else {
        printf("Running on board %d\n", boardNum);
    }

    init_tests(boardNum);
//    test_convolve(32, 8, true);
//    test_convolve2(32, 25, true);
//    test_convolve3(31, 8, true);

//    test_rot180(7, true);
//    test_padZeros(25, 3, false);
//    test_subsample(32, 4);
//    test_supersample(32, 4, true);
//    test_gridToMatrix(25, 5);
//    test_matrixToGrid(25, 5);
//    test_localMax(32, 4);
//    test_localSum(32, 4);
    test_sampleMultinomial(49);
}
